#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>

// CUDA kernel: each thread adds one element
__global__ void vectorAdd(const float *A, const float *B, float *C, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        C[i] = A[i] + B[i];
    }
}

int main() {
  int N = 1 << 16;  // 65536 elements
  size_t size = N * sizeof(float);

  // Allocate host memory
  std::vector<float> h_A(N, 1.0f); // all 1s
  std::vector<float> h_B(N, 2.0f); // all 2s
  std::vector<float> h_C(N);

  // Allocate device memory
  float *d_A, *d_B, *d_C;
  hipMalloc((void**)&d_A, size);
  hipMalloc((void**)&d_B, size);
  hipMalloc((void**)&d_C, size);

  // Copy input data to device
  hipMemcpy(d_A, h_A.data(), size, hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B.data(), size, hipMemcpyHostToDevice);

  // Launch kernel with enough blocks/threads
  int threadsPerBlock = 256;
  int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
  vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

  // Copy result back to host
  hipMemcpy(h_C.data(), d_C, size, hipMemcpyDeviceToHost);

  // Verify result
  bool success = true;
  for (int i = 0; i < N; i++) {
    if (h_C[i] != 3.0f) {
      success = false;
      break;
    }
  }

  std::cout << (success ? "PASS" : "FAIL") << std::endl;

  // Cleanup
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);

  return 0;
}

